
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

// Add all of the elements of the two arrays together using the device (GPU)
__global__ void add(int n, float *x, float *y) {

  // Calculate the starting value for the for loop's index
  int index = ( blockDim.x * blockIdx.x ) + threadIdx.x;

  // Calculate the stride between elements of the arrays
  int stride = blockDim.x * gridDim.x;

  // Add the elements from array x and array y within the block
  for (int i = index; i < n; i += stride)
    // Store the result in y
    y[i] = x[i] + y[i];
}

int main(void) {

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // The number of elements in each array (1,048,576)
  int num_of_elements = 1<<20;

  // The total memory size for one array of the float data type
  int size = num_of_elements * sizeof(float);

  // Pointers (float) for the arrays
  float* x = NULL;
  float* y = NULL;

  // Print some intro. messages using the host (CPU)
  printf("Hello, world from CUDA C!\n");
  printf("Each array will have %d elements\n", num_of_elements);

  // Allocate memory for array x in the Unified Memory (shared memory between
  // the host and device)
  err = hipMallocManaged(&x, size);

  // Display the error message (if applicable)
  if ( err != hipSuccess) {
    printf("Failed to allocate memory for array x in the unified memory!\n");
    fprintf( stderr, "Error code: %s\n", hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  // Allocate memory for array y in the Unified Memory
  err = hipMallocManaged(&y, size);

  // Display the error message (if applicable)
  if ( err != hipSuccess) {
    printf("Failed to allocate memory for array y in the unified memory!\n");
    fprintf( stderr, "Error code: %s\n", hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  // Initialize the arrays on the host
  for ( int i = 0; i < num_of_elements; i++ ) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Tell the user we are launching the kernel
  printf("Launching the CUDA kernel\n");

  // The number of threads per block
  int block_size = 256;

  // The number of blocks in the grid
  int grid_size = (num_of_elements + block_size - 1)/(block_size);

  // Run the GPU kernel
  add<<<grid_size, block_size>>>(num_of_elements, x, y);

  // Get the status message from the device
  err = hipGetLastError();

  // Display the error message (if applicable)
  if ( err != hipSuccess ) {
    printf("Failed to launch CUDA kernel\n");
    fprintf( stderr, "Error code: %s\n", hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  // Wait for GPU to finish before continuing on the host
  hipDeviceSynchronize();

  // Check for errors (all of the values should equal 3.0f)
  float maxError = 0.0f;

  // Calculate the max. error (all of the sums were stored in the array y)
  for (int i = 0; i < num_of_elements; i++) {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }

  // Display the max. error
  printf("Max error: %f\n", maxError);

  // Free the Unified memory
  hipFree(x);
  hipFree(y);

  return 0;
}
