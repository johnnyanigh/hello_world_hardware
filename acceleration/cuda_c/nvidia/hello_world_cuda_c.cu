
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// For puts and fprintf.
#include <stdio.h>

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {

  int index = ( blockDim.x * blockIdx.x ) + threadIdx.x;

  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void) {

  // Print a message.
  puts("Hello, world from CUDA C!");

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // The number of elements in each array.
  int num_of_elements = 1<<20;

  // The size in memory of the array of floats.
  int size = num_of_elements * sizeof(float);

  // Print a message.
  printf("Each array will have %d elements\n", num_of_elements);

  // Allocate memory for the arrays in the Unified Memory.
  float *x = NULL;
  float *y = NULL;

  err = hipMallocManaged(&x, size);

  // Display the error message.
  if ( err != hipSuccess) {
    puts("Failed to allocate memory for array x in the unified memory!");
    fprintf( stderr, "Error code: %s\n", hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  err = hipMallocManaged(&y, size);

  // Display the error message.
  if ( err != hipSuccess) {
    puts("Failed to allocate memory for array y in the unified memory!");
    fprintf( stderr, "Error code: %s\n", hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  // initialize x and y arrays on the host
  for ( int i = 0; i < num_of_elements; i++ ) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  puts("Launching CUDA kernel");

  // The number of threads per block.
  int block_size = 256;

  int grid_size = ( num_of_elements + block_size - 1) / (block_size);

  // Run kernel on 1M elements on the GPU.
  add<<<grid_size, block_size>>>(num_of_elements, x, y);

  err = hipGetLastError();

  if ( err != hipSuccess ) {
    puts("Failed to launch CUDA kernel");
    fprintf( stderr, "Error code: %s\n", hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;

  for (int i = 0; i < num_of_elements; i++) {
    maxError = fmax( maxError, fabs( y[i]-3.0f ) );
  }

  std::cout << "Max error: " << maxError << std::endl;

  // Free the unified memory.
  hipFree(x);
  hipFree(y);

  return 0;
}
